#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>

#include "messages.h"
#include "usernames.h"
#include "searchKeywords.h"

#define N 25000 //num of transaction
#define THREADS_PER_BLOCK 85 //num of search words
#define NUM_CORES 4992

//TODO
// what needs to be initialized in GPU (messages, searchwords,booolean)
// what needs to be initialized in CPU (messages, searchwords,booolean)
// what needs to be cudaMemCpy and what is the type --  Copy CPU to GPU??
// filerByFood parameter types - char arrays, and bool array
// what to send to filterByFood

__global__ void filterByFood(gpu_m,gpu_s) {
    int messageIndex = blockIdx.x;
    int searchWordIndex = threadIdx.x;

    if (searchWords[searchWordIndex] == messages[messageIndex]) {
        boolResults[messageIndex] = true;
    }
}
int main(){
    __constant__ gpuMessages = messages;
    __constant__ gpuUsernames = usernames;
    __constant__ gpuSearchKeywords= searchKeywords;


    //(2) Allocate GPU memory
    char∗ gpu_m;
    char∗ gpu_s;
    // TODO: fill with false
    bool* boolResults = []

    //(3) Copy CPU to GPU
    hipMemcpy( gpu_m, gpuMessages, N∗sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy( gpu_s,gpuSearchKeywords, N∗sizeof(char), hipMemcpyHostToDevice);

    / / (4) Run GPU code

     int N = 2500
	 filterByFood<<<N,THREADS_PER_BLOCK>>>(gpu_m,gpu_s);

}