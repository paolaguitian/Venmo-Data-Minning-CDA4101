#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>

#include "CPU_messages.h"
#include "GPU_messages.h"
#include "usernames.h"
#include "searchKeywords.h"

#define N 25000 //num of transaction
#define THREADS_PER_BLOCK 85 //num of search words
#define NUM_CORES 4992

__global__ void filterByFood(bool* boolResults) {
    int messageIndex = blockIdx.x;
    int searchWordIndex = threadIdx.x;

    while(*GPU_messages[messageIndex]) {
        char* currentMessage = GPU_messages[messageIndex]; //begin str
        char* criteria = GPU_searchKeywords[searchWordIndex]; // pattern substr

        while(*GPU_messages[messageIndex] && *criteria && *GPU_messages[messageIndex] == *criteria ) {
            GPU_messages[messageIndex]++;
            criteria++;
        }

        if(!*criteria)
            boolResults[messageIndex] = true;

        GPU_messages[messageIndex] = currentMessage + 1;
    }
}

int main(){
    FILE *fp;
    fp = fopen("Output.txt", "w");

    //(1) Allocate CPU memory
    bool* boolResults = (bool*) malloc(N*sizeof(bool));

    //(2) Allocate GPU memory
    bool* GPU_boolResults;
    hipMalloc(&GPU_boolResults, N*sizeof(bool));


    // (3) Run GPU code
     filterByFood<<<N,THREADS_PER_BLOCK>>>(GPU_boolResults);
     cudaMemCpy(boolResults, GPU_boolResults, N*sizeof(bool), hipMemcpyDeviceToHost);

     for(int i = 0; i < N; i++){
         if(boolResults[i]){
             fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
         }
     }
     fclose(fp);
     hipFree(GPU_boolResults);
     free(boolResults);
     return 0;
}