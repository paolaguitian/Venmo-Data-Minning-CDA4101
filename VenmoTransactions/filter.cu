#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <sys/time.h>

#include "CPU_messages.h"
#include "GPU_messages.h"
#include "GPU_searchKeywords.h"
#include "CPU_searchKeywords.h"
#include "usernames.h"


#define N 2500 //num of transaction
#define THREADS_PER_BLOCK 124 //num of search words
#define NUM_CORES 4992

// function to filter food items within GPU
__global__ void GPU_filterByFood(bool* GPU_boolResults) {
    int numloop = N / NUM_CORES;
    if(numloop == 0) numloop = 1; 

    for(int m = 0; m < numloop; m++) {
        int messageIndex = blockIdx.x + (NUM_CORES * m);
        int criteriaIndex = threadIdx.x;
        const char *CM = GPU_messages[messageIndex]; //begin str
        const char *criteria = GPU_searchKeywords[criteriaIndex]; // pattern substr
        int s= 0;
        int i = 0;
        int j = 0;

        
        while(CM[s] != '\0') {
            i = 0, j = s;

            while(criteria[i] != '\0' && CM[j] != '\0') {
                if(CM[j] == criteria[i]) { 
                    j++;
                    i++;
                }

                else break;
            }
            if(criteria[i] == '\0'){ //match! :)
                GPU_boolResults[messageIndex] = true;
            }

            s++;
        }
    }
}

// function to filter food items within CPU
void CPU_filterByFood(bool* CPU_boolResults) { 
    int i;
    int j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < THREADS_PER_BLOCK; j++) {
            const char *CM = CPU_messages[i]; //begin str
            const char *criteria = CPU_searchKeywords[j]; // pattern substr
            int s = 0;
            int m = 0;
            int n = 0;

            while(CM[s] != '\0') {
                m = 0, n = s;

                while(criteria[m] != '\0' && CM[n] != '\0') {
                    if(criteria[m] == CM[n]) { 
                        m++;
                        n++;                
                    }

                    else break;
                }
                if(criteria[m] == '\0'){ //match! :)
                    CPU_boolResults[i] = true;
                }

                s++;
            }
        }
    }
}

int main() {
    FILE *fp;
    fp = fopen("GPU_results.txt", "w");
    struct timeval start, end;
        
    //(1) Allocate CPU memory
    bool* boolResults = (bool*) malloc(N*sizeof(bool));
    bool* CPU_boolResults = (bool*) malloc(N*sizeof(bool));

    //run  CPU code
    gettimeofday( &start, 0);
    
    CPU_filterByFood(CPU_boolResults);        

    gettimeofday( &end, 0 );
    double CPU_time = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;


    //(2) Allocate GPU memory
    bool* GPU_boolResults;
    hipMalloc(&GPU_boolResults, N*sizeof(bool));

    // (3) Run GPU code
     gettimeofday( &start, 0);

     GPU_filterByFood<<<N,THREADS_PER_BLOCK>>>(GPU_boolResults);
     hipMemcpy(boolResults, GPU_boolResults, N*sizeof(bool), hipMemcpyDeviceToHost);

     gettimeofday( &end, 0 );
     double GPU_time = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
     
     //print GPU output
     for(int i = 0; i < N; i++){
         if(boolResults[i]){
             fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
         }
     }

     //print CPU output
     fclose(fp);
     fp = fopen("CPU_results.txt", "w");

     for(int i = 0; i < N; i++){
        if(CPU_boolResults[i]){
            fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
        }
    }


    fclose(fp);

     hipFree(GPU_boolResults);
     free(boolResults);
     free(CPU_boolResults);

     printf("Number of transactions: %d\n", N);
     printf("-------------------------\n");
     printf("Time taken by CPU: %lf ms\n", CPU_time);
     printf("Time taken by GPU: %lf ms\n", GPU_time);
     return 0;
}