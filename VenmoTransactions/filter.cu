#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>

#include "CPU_messages.h"
#include "GPU_messages.h"
#include "usernames.h"
#include "GPU_searchKeywords.h"

#define N 2500 //num of transaction
#define THREADS_PER_BLOCK 124 //num of search words
#define NUM_CORES 4992

__global__ void filterByFood(bool* GPU_boolResults) {
    const char *CM = GPU_messages[blockIdx.x]; //begin str
    const char *criteria = GPU_searchKeywords[threadIdx.x]; // pattern substr
    int s= 0;
    int i = 0;
    int j = 0;

    while(CM[s] != '\0') {
        i = 0, j = s;

        while(criteria[i] != '\0' && CM[j] != '\0') {
            if(CM[j] == criteria[i]) { 
                j++;
                i++;
            }

            else break;
        }
        if(criteria[i] == '\0'){ //match! :)
            GPU_boolResults[blockIdx.x] = true;
        };

        s++;
    }
}

int main(){
    FILE *fp;
    fp = fopen("Output.txt", "w");

    //(1) Allocate CPU memory
    bool* boolResults = (bool*) malloc(N*sizeof(bool));

    //(2) Allocate GPU memory
    bool* GPU_boolResults;
    hipMalloc(&GPU_boolResults, N*sizeof(bool));


    // (3) Run GPU code
     filterByFood<<<N,THREADS_PER_BLOCK>>>(GPU_boolResults);
     hipMemcpy(boolResults, GPU_boolResults, N*sizeof(bool), hipMemcpyDeviceToHost);

     for(int i = 0; i < N; i++){
         if(boolResults[i]){
             fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
         }
     }
     fclose(fp);
     hipFree(GPU_boolResults);
     free(boolResults);
     return 0;
}