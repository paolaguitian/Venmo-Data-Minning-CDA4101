#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <time.h>

#include "CPU_messages.h"
#include "GPU_messages.h"
#include "GPU_searchKeywords.h"
#include "CPU_searchKeywords.h"
#include "usernames.h"


#define N 2500 //num of transaction
#define THREADS_PER_BLOCK 124 //num of search words
#define NUM_CORES 4992

__global__ void GPU_filterByFood(bool* GPU_boolResults) {
    const char *CM = GPU_messages[blockIdx.x]; //begin str
    const char *criteria = GPU_searchKeywords[threadIdx.x]; // pattern substr
    int s= 0;
    int i = 0;
    int j = 0;

    while(CM[s] != '\0') {
        i = 0, j = s;

        while(criteria[i] != '\0' && CM[j] != '\0') {
            if(CM[j] == criteria[i]) { 
                j++;
                i++;
            }

            else break;
        }
        if(criteria[i] == '\0'){ //match! :)
            GPU_boolResults[blockIdx.x] = true;
        }

        s++;
    }
}

void CPU_filterByFood(bool* CPU_boolResults) { 
    int i;
    int j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < THREADS_PER_BLOCK; j++) {
            const char *CM = CPU_messages[i]; //begin str
            const char *criteria = CPU_searchKeywords[j]; // pattern substr
            int s = 0;
            int m = 0;
            int n = 0;

            while(CM[s] != '\0') {
                m = 0, n = s;

                while(criteria[m] != '\0' && CM[n] != '\0') {
                    if(criteria[m] == CM[n]) { 
                        m++;
                        n++;                
                    }

                    else break;
                }
                if(criteria[m] == '\0'){ //match! :)
                    CPU_boolResults[i] = true;
                }

                s++;
            }
        }
    }
}

int main() {
    FILE *fp;
    fp = fopen("GPU_results.txt", "w");
    clock_t start, end;
        
    //(1) Allocate CPU memory
    bool* boolResults = (bool*) malloc(N*sizeof(bool));
    bool* CPU_boolResults = (bool*) malloc(N*sizeof(bool));

    //run  CPU code
    start = clock();
    
    CPU_filterByFood(CPU_boolResults);        

    end = clock();
    double CPU_time = double(end - start) / double(CLOCKS_PER_SEC);

    //(2) Allocate GPU memory
    bool* GPU_boolResults;
    hipMalloc(&GPU_boolResults, N*sizeof(bool));

    // (3) Run GPU code
     start = clock();

     GPU_filterByFood<<<N,THREADS_PER_BLOCK>>>(GPU_boolResults);
     hipMemcpy(boolResults, GPU_boolResults, N*sizeof(bool), hipMemcpyDeviceToHost);

     end = clock();
     double GPU_time = double(end - start) / double(CLOCKS_PER_SEC);
     //print GPU output
     for(int i = 0; i < N; i++){
         if(boolResults[i]){
             fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
         }
     }

     //print CPU output
     fclose(fp);
     fp = fopen("CPU_results.txt", "w");

     for(int i = 0; i < N; i++){
        if(CPU_boolResults[i]){
            fprintf(fp,"User: %s Message: %s\n", usernames[i], CPU_messages[i]);
        }
    }


    fclose(fp);

     hipFree(GPU_boolResults);
     free(boolResults);
     free(CPU_boolResults);

     printf("Time taken by CPU: %lf\n", CPU_time);
     printf("Time taken by GPU: %lf\n", GPU_time);
     return 0;
}